#include <hip/hip_runtime.h>
#include <hipsparse.h>         // cusparseSpMM

#include <nvtx3/nvToolsExt.h>
#define IMPROVE 0

#define CUDA_BLOCKDIM 1024

#define WARP_SIZE 32



//nvcc -Xcompiler -fPIC -shared -lcusparse -o block_attn_mask.so block_attn_mask_warp.cu

//dAttn, dMask, dSum_mat, dOffsets, dColumns, nnz, seq_len, num_batches, scale, i

static __device__ __inline__ uint32_t __get_warpid(){
  uint32_t warpid;
  asm volatile("mov.u32 %0, %%warpid;" : "=r"(warpid));
  return warpid;}

static __device__ __inline__ uint32_t __get_laneid(){
  uint32_t laneid;
  asm volatile("mov.u32 %0, %%laneid;" : "=r"(laneid));
  return laneid;}


__global__ void scale_softmax_kernel_large(float *d_values, float *d_mask, int *dSum_mat, int *dOffsets, int *dColumns, int nnz, int seq_len, int num_batches, float scale, int tmp)
{
    // coalescing issue
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;

    if (i >= seq_len*num_batches)
        return;
    
    for(int t = 0; t < tmp; t++){
        //int tid = (threadIdx.x * tmp) + t; //0~4095
        int tid = (t * blockDim.x) + threadIdx.x; //0~4095
        int block_size = dSum_mat[tid];
        int block_idx = bid*nnz + dOffsets[tid];

        float max = -100.0;
        for (int k = 0; k < block_size; k++){
            d_values[block_idx + k] = d_values[block_idx + k] * scale - d_mask[blockIdx.x*seq_len + dColumns[block_idx + k]];
            if (max < d_values[block_idx + k]){
                max = d_values[block_idx + k];
            }
        }
        float sum = 0.0f;
        for (int k = 0; k < block_size; k++){
            sum += expf(d_values[block_idx + k] - max);

        }
        sum += (seq_len - block_size) * expf(-max);
        sum = 1/sum;
        for (int k = 0; k < block_size; k++){
            if(sum == 0){
                d_values[block_idx + k] = 0;
            }
            else{
                d_values[block_idx + k] = expf(d_values[block_idx + k] - max) * sum;
            }
        }
    }
}


__inline__ __device__
float warp_reduce_max(float value){
    for(int w = 1; w < WARP_SIZE; w = w << 1){
        float tmp = __shfl_xor_sync(0xffffffff, value, w);
        value = fmaxf(value, tmp);
    }
    return value;
}
__inline__ __device__
float warp_reduce_sum(float value){
    float ret = value;

    for(int w = 1; w < WARP_SIZE; w = w << 1){
        float tmp = __shfl_xor_sync(0xffffffff, ret, w);
        ret += tmp;
    }
    return ret;
}

__global__ void scale_softmax_kernel(float *d_values, float *d_mask, int *dSum_mat, int *dOffsets, int *dColumns, int nnz, int seq_len, int num_batches, float scale)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // 0 ~ seq_len * num_batch * WARP_SIZE
    

    if(i > WARP_SIZE *seq_len * num_batches)
        return;
    
    int warp_id = i / WARP_SIZE; // 0 ~ seq_len * num_batch
    int lane = i % WARP_SIZE;
    int batch_id = warp_id / seq_len;
    int row_to_compute = warp_id % seq_len;//(t * blockDim.x) + threadIdx.x; //0~4095
    
    int block_size = dSum_mat[row_to_compute];
    int block_idx = batch_id *nnz + dOffsets[row_to_compute];


    float max1 = 0.0;
    int mask_tmp = batch_id*seq_len;
    float value_tmp;
    
    for (int k = block_idx + lane; k < block_idx + block_size; k+=WARP_SIZE){
        value_tmp = d_values[k] * scale - d_mask[mask_tmp + dColumns[k]];
        
        if (max1 < value_tmp){
            max1 = value_tmp;
        }
        d_values[k] = value_tmp;
        
    }

    float max = warp_reduce_max(max1);

    float sum = 0.0f;
    for (int k = block_idx + lane; k < block_idx + block_size; k+= WARP_SIZE){
        sum += expf(d_values[k] - max);
    }
    sum = warp_reduce_sum(sum);
    sum += (seq_len - block_size) * expf(-max);
    sum = 1/sum;

    if(sum == 0){
        for (int k = block_idx + lane; k < block_idx + block_size; k+= WARP_SIZE){
            d_values[k] = 0;
        }    
    }
    else{
        for (int k = block_idx + lane; k < block_idx + block_size; k+= WARP_SIZE){
            d_values[k] = expf(d_values[k] - max) * sum;
        } 
    }
    
}

__global__ void softmax_scale_backward_kernel(float *dGradAttnScore, float *dAttnScore,
                                            float *dGradAttn, int *dSum_mat, int *dOffsets, int nnz, int seq_len, int num_batches, float scale)
{
    // coalescing issue
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= WARP_SIZE * seq_len*num_batches)
        return;

    int warp_id = i / WARP_SIZE; // 0 ~ seq_len * num_batch
    int lane = i % WARP_SIZE;
    int batch_id = warp_id / seq_len;
    int row_to_compute = warp_id % seq_len;//(t * blockDim.x) + threadIdx.x; //0~4095
    
    int block_size = dSum_mat[row_to_compute];
    int block_idx = batch_id *nnz + dOffsets[row_to_compute];




    float grad_sum = 0.0f;
    for (int k = block_idx  + lane ; k < block_idx + block_size; k += WARP_SIZE){
        float val_tmp = dGradAttnScore[k] * dAttnScore[k];
        dGradAttnScore[k] = val_tmp;
        grad_sum += val_tmp;
    }
    grad_sum = warp_reduce_sum(grad_sum);
    
    for (int k = block_idx + lane; k < block_idx + block_size; k+= WARP_SIZE){
        dGradAttn[k] = (dGradAttnScore[k] - (dAttnScore[k] * grad_sum)) * scale;
    }
}

__global__ void softmax_scale_backward_kernel_large(float *dGradAttnScore, float *dAttnScore, 
                                            float *dGradAttn, int *dSum_mat, int *dOffsets, int nnz, int seq_len, int num_batches, float scale, int tmp)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int bid = blockIdx.x;

    if (i >= seq_len*num_batches)
        return;

    for(int t = 0; t < tmp; t++){
        //int tid = (threadIdx.x * tmp) + t; //0~4095
        int tid = (t * blockDim.x) + threadIdx.x; //0~4095
        int block_size = dSum_mat[tid];
        int block_idx = bid*nnz + dOffsets[tid];

        float grad_sum = 0.0f;
        for (int k = 0; k < block_size; k++){
            dGradAttnScore[block_idx + k] = dGradAttnScore[block_idx + k] * dAttnScore[block_idx + k];
            grad_sum += dGradAttnScore[block_idx + k];
        }
        
        // for (int k = 0; k < block_size; k++){
        //     grad_sum += dGradAttnScore[block_idx + k];
        // }
        
        for (int k = 0; k < block_size; k++){
            dGradAttn[block_idx + k] = (dGradAttnScore[block_idx + k] - (dAttnScore[block_idx + k] * grad_sum)) * scale;
        }
    }
}

struct SddmmOperation
{
    void *sddmm_dBuffer = NULL;
    hipsparseHandle_t sddmm_handle = NULL;
    SddmmOperation()
    {
    }
    ~SddmmOperation()
    {
    }
    void prepare_resources(){

    }
    void sddmm(hipsparseHandle_t handle, float *dQuery, float *dKey, float *dAttn, int *d_offsets, int *d_columns, int seq_len, int emb_dim, int nnz, int num_batches)
    {

        size_t bufferSize = 0;
        int lda = emb_dim;
        int ldb = seq_len;
        int input_size = seq_len * emb_dim;

        float alpha = 1.0f;
        float beta = 0.0f;

        if (sddmm_handle == NULL)
        {
        hipsparseCreate(&sddmm_handle);
        }

        hipsparseDnMatDescr_t matA, matB;
        hipsparseSpMatDescr_t matC;
        nvtxRangePush("SDDMM prepare resources");

        // Create dense matrix A
        hipsparseCreateDnMat(&matA, seq_len, emb_dim, lda, dQuery,
                            HIP_R_32F, HIPSPARSE_ORDER_ROW);
        hipsparseDnMatSetStridedBatch(matA, num_batches, input_size);
        // Create dense matrix B
        hipsparseCreateDnMat(&matB, emb_dim, seq_len, ldb, dKey,
                            HIP_R_32F, HIPSPARSE_ORDER_ROW);
        hipsparseDnMatSetStridedBatch(matB, num_batches, input_size);
        // Create sparse matrix C in CSR format
        hipsparseCreateCsr(&matC, seq_len, seq_len, nnz,
                          d_offsets, d_columns, dAttn,
                          HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                          HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
        hipsparseCsrSetStridedBatch(matC, num_batches, 0, nnz);
        // allocate an external buffer if needed
        
        nvtxRangePush("SDDMM buffer size");
        hipsparseSDDMM_bufferSize(
            sddmm_handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC, HIP_R_32F,
            HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize);

        nvtxRangePop();

        nvtxRangePop();
        nvtxRangePush("Malloc from sddmm");
        if (sddmm_dBuffer == NULL)
        {
            //printf("MALLOC from SDDMM %d %d %X\n", getpid(), gettid(), this);
            hipMalloc(&sddmm_dBuffer, bufferSize);
        }
        nvtxRangePop();

        nvtxRangePush("SDDMM PREPROCESS");
        // execute preprocess (optional)
        hipsparseSDDMM_preprocess(
            sddmm_handle,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &alpha, matA, matB, &beta, matC, HIP_R_32F,
            HIPSPARSE_SDDMM_ALG_DEFAULT, sddmm_dBuffer);
        nvtxRangePop();

        nvtxRangePush("SDDMM COMPUTE");
        hipsparseSDDMM(sddmm_handle,
                      HIPSPARSE_OPERATION_NON_TRANSPOSE,
                      HIPSPARSE_OPERATION_NON_TRANSPOSE,
                      &alpha, matA, matB, &beta, matC, HIP_R_32F,
                      HIPSPARSE_SDDMM_ALG_DEFAULT, sddmm_dBuffer);
        // destroy matrix/vector descriptors
        nvtxRangePop();
        nvtxRangePush("nvtx DESTROY Resources");
        hipsparseDestroyDnMat(matA);
        hipsparseDestroyDnMat(matB);
        hipsparseDestroySpMat(matC);
        // cudaFree(sddmm_dBuffer);
        nvtxRangePop();
    }
};
struct SpmmOperation
{
    void *spmm_dBuffer = NULL;
    hipsparseHandle_t spmm_handle = NULL;
    SpmmOperation()
    {
    }
    ~SpmmOperation()
    {
    }
    void spmm(hipsparseHandle_t handle, hipsparseOperation_t opA, void *dBuffer, float *dA, float *dB, float *dC, int *d_offsets, int *d_columns, int seq_len, int emb_dim, int nnz, int num_batches)
    {
        // Host problem definition
        int ldb = emb_dim;
        int ldc = emb_dim;
        int output_size = seq_len * emb_dim;

        float alpha = 1.0f;
        float beta = 0.0f;
        // float milliseconds = 0;

        hipsparseSpMatDescr_t matA;
        hipsparseDnMatDescr_t matB, matC;

        size_t bufferSize = 0;

        if (spmm_handle == NULL)
        {
        hipsparseCreate(&spmm_handle);
        }
        

        // cudaEvent_t start, stop;
        // cudaEventCreate(&start);
        // cudaEventCreate(&stop);
        // float milliseconds = 0;

        // cudaEventRecord(start);
        nvtxRangePush("SPMM resource prepare");
        hipsparseCreateCsr(&matA, seq_len, seq_len, nnz,
                          d_offsets, d_columns, dA,
                          HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                          HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("cusparseCreateCsr A : %f\n",milliseconds);

        // cudaEventRecord(start);
        hipsparseCsrSetStridedBatch(matA, num_batches, 0, nnz);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("cusparseCsrSetStridedBatch A : %f\n",milliseconds);

        // cudaEventRecord(start);
        hipsparseCreateDnMat(&matB, seq_len, emb_dim, ldb, dB,
                            HIP_R_32F, HIPSPARSE_ORDER_ROW);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("cusparseCreateDnMat B : %f\n",milliseconds);

        // cudaEventRecord(start);
        hipsparseDnMatSetStridedBatch(matB, num_batches, output_size);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("cusparseDnMatSetStridedBatch B : %f\n",milliseconds);

        // cudaEventRecord(start);
        hipsparseCreateDnMat(&matC, seq_len, emb_dim, ldc, dC,
                            HIP_R_32F, HIPSPARSE_ORDER_ROW);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("cusparseCreateDnMat C : %f\n",milliseconds);

        // cudaEventRecord(start);
        hipsparseDnMatSetStridedBatch(matC, num_batches, output_size);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("cusparseDnMatSetStridedBatch C : %f\n",milliseconds);

        // cudaEventRecord(start);
        nvtxRangePush("SPMM BUFFER SIZE");
        hipsparseSpMM_bufferSize(spmm_handle,
                                opA, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                HIPSPARSE_SPMM_CSR_ALG2, &bufferSize);
        nvtxRangePop();
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("cusparseSpMM_bufferSize dBuffer : %f\n",milliseconds);

        // cudaEventRecord(start);

        nvtxRangePop();
        nvtxRangePush("SPMM Malloc");
#if IMPROVE == 0
        if (dBuffer == NULL)
        {
            //printf("MALLOC from SPMM %d %d %X\n", getpid(), gettid(), this);
            hipMalloc(&dBuffer, bufferSize);
        }
#else
        if (spmm_dBuffer == NULL)
        {
            //printf("MALLOC from SPMM %d %d %X\n", getpid(), gettid(), this);
            cudaMalloc(&spmm_dBuffer, bufferSize);
        }

#endif
        nvtxRangePop();
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("cudaMalloc dBuffer : %f\n",milliseconds);

        // nvtxRangePush("SpMM PREPROCESS");
        // // execute preprocess (optional)
        // cusparseSpMM_preprocess(
        //     spmm_handle,
        //     opA,
        //     CUSPARSE_OPERATION_NON_TRANSPOSE,
        //     &alpha, matA, matB, &beta, matC, CUDA_R_32F,
        //     CUSPARSE_SPMM_CSR_ALG2, spmm_dBuffer);
        // nvtxRangePop();

        // cudaEventRecord(start);
        nvtxRangePush("SPMM Compute");
#if IMPROVE == 0
        hipsparseSpMM(spmm_handle,
                     opA, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                     &alpha, matA, matB, &beta, matC, HIP_R_32F,
                     HIPSPARSE_SPMM_CSR_ALG2, dBuffer);
#else
        cusparseSpMM(spmm_handle,
                     opA, CUSPARSE_OPERATION_NON_TRANSPOSE,
                     &alpha, matA, matB, &beta, matC, CUDA_R_32F,
                     CUSPARSE_SPMM_CSR_ALG2, spmm_dBuffer);
        
#endif
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("cusparseSpMM : %f\n",milliseconds);
        nvtxRangePop();
        nvtxRangePush("SPMM Resource Free");
        hipsparseDestroySpMat(matA);
        hipsparseDestroyDnMat(matB);
        hipsparseDestroyDnMat(matC);
#if IMPROVE==0
        hipFree(dBuffer);
#endif
        nvtxRangePop();
    }
};


struct Attention{

    hipsparseHandle_t     handle = NULL;
    void* dBuffer    = NULL;
    hipEvent_t start, stop;
    float milliseconds = 0;
    float scale = 0;
    int emb_dim, seq_len, num_batches, block_size, tmp_seq_len, tmp_num_batches, tmp, grid_size, blockdim_size;

    SddmmOperation *sd1, *sd2;
    SpmmOperation *sp1, *sp2, *sp3, *sp4;

    Attention(int hemb_dim, int hseq_len, int hnum_batches, int hblock_size)
    {
        scale = 1/sqrtf(float(hemb_dim));
        emb_dim = hemb_dim;
        seq_len = hseq_len;
        num_batches = hnum_batches;
        block_size = hblock_size;

        tmp_seq_len = 1024;

        if(seq_len>=tmp_seq_len){
            tmp = (int)(seq_len/tmp_seq_len);
            tmp_num_batches = num_batches * tmp;
                        
        }
        else{
            tmp_seq_len = seq_len;
            tmp_num_batches = num_batches;
            tmp = 0;
        }

        blockdim_size = CUDA_BLOCKDIM > seq_len ? seq_len : CUDA_BLOCKDIM;
        grid_size = (((seq_len * WARP_SIZE) + blockdim_size - 1) / blockdim_size) * num_batches;

        hipsparseCreate(&handle);

        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        sd1 = new SddmmOperation();
        sd2 = new SddmmOperation();
        sp1 = new SpmmOperation();
        sp2 = new SpmmOperation();
        sp3 = new SpmmOperation();
        sp4 = new SpmmOperation();
    }

    ~Attention(){
        delete sd1;
        delete sd2;
        delete sp1;
        delete sp2;
        delete sp3;
        delete sp4;
        hipsparseDestroy(handle);
    }

    void attn_forward(float *dQuery, float *dKey, float *dValue, float *dMask, float *dAttn, float *dOut, int *dOffsets, int *dColumns, int *dSum_mat, int nnz)
    {
        //cudaEventRecord(start);
        sd1->sddmm(handle, dQuery, dKey, dAttn, dOffsets, dColumns, seq_len, emb_dim, nnz, num_batches);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("SDDMM : %f\n",milliseconds);

        // cudaEventRecord(start);
        //scale_softmax_kernel<<<num_batches, seq_len>>>(dAttnScore, dAttn, dMask, dSum_mat, dOffsets, dColumns, nnz, seq_len, num_batches, scale);
        // printf("tmp : %d\n",tmp);
        // printf("tmp_num_batches : %d\n",tmp_num_batches);
        // printf("tmp_seq_len : %d\n",tmp_seq_len);
        // printf("num_batches : %d\n",num_batches);
        // cudaEventRecord(start);
        
        // float* before;
        // float* host_dAttn, *host_before;
        // cudaMalloc(&before, sizeof(float) * nnz * num_batches);
        // host_dAttn = (float*)malloc(sizeof(float) * nnz * num_batches);
        // host_before = (float*)malloc(sizeof(float) * nnz * num_batches);
        // cudaMemcpy(before, dAttn, sizeof(float) * nnz * num_batches, cudaMemcpyDeviceToDevice);


        // scale_softmax_kernel_large<<<num_batches, tmp_seq_len>>>(before, dMask, dSum_mat, dOffsets, dColumns, nnz, seq_len, num_batches, scale, tmp);
        
        //cudaEventRecord(start);
        scale_softmax_kernel<<<grid_size, blockdim_size>>>(dAttn, dMask, dSum_mat, dOffsets, dColumns, nnz, seq_len, num_batches, scale);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("softmax : %f\n",milliseconds);



        // cudaDeviceSynchronize();

        // cudaMemcpy(host_dAttn, dAttn, sizeof(float) * nnz * num_batches, cudaMemcpyDeviceToHost);
        // cudaMemcpy(host_before, before, sizeof(float) * nnz * num_batches, cudaMemcpyDeviceToHost);

        // cudaDeviceSynchronize();
        // for (int i =0 ;i< nnz * num_batches; i++)
        //     {
        //         if (host_before[i] != host_dAttn[i])
        //             printf("AFTER KERNEL LAUNCH!!! %.20f\n", abs(host_before[i] - host_dAttn[i]));
        //     }
        // cudaFree(before);
        // free(host_before);
        // free(host_dAttn);

        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("Forward scale_softmax_function : %f\n",milliseconds);

        //cudaEventRecord(start);
        sp1->spmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, dBuffer, dAttn, dValue, dOut, dOffsets, dColumns, seq_len, emb_dim, nnz, num_batches);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("SPMM : %f\n",milliseconds);
    }

    void attn_backward(float *dQuery, float *dKey, float *dValue, float *dAttnScore, float *dGradOutput, float *dGradAttnScore, 
                    float *dGradAttn, float *dGradQuery, float *dGradKey, float *dGradValue,
                    int *dOffsets, int *dColumns, int *dSum_mat, int nnz)
    {

        // cudaEventRecord(start);
        sd1->sddmm(handle, dGradOutput, dValue, dGradAttnScore, dOffsets, dColumns, seq_len, emb_dim, nnz, num_batches);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("Backward SDDMM : %f\n",milliseconds);

        // cudaEventRecord(start);
        // softmax_scale_backward_kernel<<<num_batches, seq_len>>>(dGradAttnScore, dAttnScore, dGradAttnScoreScale, dGradAttnScale, dGradAttn, dSum_mat, dOffsets, nnz, seq_len, num_batches, scale);
        
        
        //softmax_scale_backward_kernel_large<<<num_batches, tmp_seq_len>>>(dGradAttnScore, dAttnScore, dGradAttn, dSum_mat, dOffsets, nnz, seq_len, num_batches, scale, tmp);
       
        softmax_scale_backward_kernel<<<grid_size, blockdim_size>>>(dGradAttnScore, dAttnScore, dGradAttn, dSum_mat, dOffsets, nnz, seq_len, num_batches, scale);
        
  
        
        
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("Backward softmax_scale_backward_function : %f\n",milliseconds);

        // cudaEventRecord(start);
        sp2->spmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, dBuffer, dGradAttn, dKey, dGradQuery, dOffsets, dColumns, seq_len, emb_dim, nnz, num_batches);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("Backward SPMM 1 : %f\n",milliseconds);

        // cudaEventRecord(start);
        sp3->spmm(handle, HIPSPARSE_OPERATION_TRANSPOSE, dBuffer, dGradAttn, dQuery, dGradKey, dOffsets, dColumns, seq_len, emb_dim, nnz, num_batches);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("Backward SPMM 2 : %f\n",milliseconds);

        // cudaEventRecord(start);
        sp4->spmm(handle, HIPSPARSE_OPERATION_TRANSPOSE, dBuffer, dAttnScore, dGradOutput, dGradValue, dOffsets, dColumns, seq_len, emb_dim, nnz, num_batches);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("Backward SPMM 3 : %f\n",milliseconds);

    }
};

extern "C" Attention *init(int emb_dim, int seq_len, int num_batches, int block_size)
{
    return new Attention(emb_dim, seq_len, num_batches, block_size);
}

extern "C" void attn_forward(Attention* attn, float *hQuery, float *hKey, float *hValue, float *hMask, float *hAttn, float *hOut, int *hOffsets, 
                    int *hColumns, int *hSum_mat, int nnz){
    attn->attn_forward(hQuery, hKey, hValue, hMask, hAttn, hOut, hOffsets, hColumns, hSum_mat, nnz);
}

extern "C" void attn_backward(Attention* attn, float *hQuery, float *hKey, float *hValue, float *hAttnScore, float *hGradOutput, float *hGradAttnScore, 
                    float *hGradAttn, float *hGradQuery, float *hGradKey, float *hGradValue,
                    int *hOffsets, int *hColumns, int *hSum_mat, int nnz){
    attn->attn_backward(hQuery, hKey, hValue, hAttnScore, hGradOutput, hGradAttnScore, hGradAttn, hGradQuery, hGradKey, hGradValue, 
                        hOffsets, hColumns, hSum_mat, nnz);
}

extern "C" void destroy(Attention* attn) {
    delete attn;
}
