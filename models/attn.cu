
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> 
#include <hipblas.h>

#include <math.h>
#include <stdio.h>

//nvcc -Xcompiler -fPIC -shared -lcublas -o attn.so attn.cu

__global__ void max_sum_exp_kernel(float *d_values, int seq_len, float scale, int num_batches)
{
    int i = blockIdx.x * blockDim.x * seq_len + threadIdx.x;
    //printf("%d\n", i);
    //float scale = sqrtf(float(emb_dim));
    if (i >= seq_len*seq_len*num_batches)
        return;

    for (int k = 0; k < seq_len; k++){
        d_values[seq_len * k + i] = d_values[seq_len * k + i] * scale;
    }

    float max = -10.0;
    for (int k = 0; k < seq_len; k++){
        if (max < d_values[seq_len * k + i]){
            max = d_values[seq_len * k + i];
        }
    }

    float sum = 0.0f;
    for (int k = 0; k < seq_len; k++){
        sum += expf(d_values[seq_len * k + i] - max);
    }
    sum = 1/sum;
    for (int k = 0; k < seq_len; k++){
        d_values[seq_len * k + i] = expf(d_values[seq_len * k + i] - max) * sum;
    }
}

__global__ void max_sum_exp_kernel_large(float *d_values, int seq_len, float scale, int num_batches, int tmp)
{
    //int i = blockIdx.x * seq_len * seq_len + (threadIdx.x * tmp + idx);
    int bid = blockIdx.x * seq_len; //blockDim.x : 1024
    //printf("%d\n", i);
    for(int t = 0; t<tmp; t++){
        int i = blockIdx.x * seq_len * seq_len + (t * blockDim.x) + threadIdx.x;
        if (i >= seq_len*seq_len*num_batches)
            return;
        //printf("%d\n", bid);
        float max = -10.0;
        for (int k = 0; k < seq_len; k++){
            d_values[seq_len * k + i] = (d_values[seq_len * k + i] * scale);
            if (max < d_values[seq_len * k + i]){
                max = d_values[seq_len * k + i];
            }
        }

        float sum = 0.0f;
        for (int k = 0; k < seq_len; k++){
            sum += expf(d_values[seq_len * k + i] - max);
        }

        for (int k = 0; k < seq_len; k++){
            d_values[seq_len * k + i] = expf(d_values[seq_len * k + i] - max) / sum;
        }
    }
}

__global__ void softmax_scale_backward_kernel(float *dGradAttnScore, float *dAttnScore, float *dGradAttnScale,
                                            float *dGradAttn, int seq_len, int emb_dim, int num_batches, float scale)
{
    int i = blockIdx.x * blockDim.x * seq_len + threadIdx.x;

    if (i >= seq_len*seq_len*num_batches)
        return;
    
    for (int k = 0; k < seq_len; k++){
        dGradAttnScore[k * seq_len + i] = dGradAttnScore[k * seq_len + i] * dAttnScore[k * seq_len + i];
    }

    float sum = 0.0f;
    for (int k = 0; k < seq_len; k++){
        sum += dGradAttnScore[k * seq_len + i];

    }
    for (int k = 0; k < seq_len; k++){
        dGradAttn[k * seq_len + i] = (dGradAttnScore[k * seq_len + i] - (dAttnScore[k * seq_len + i] * sum)) * scale;
    }
}

__global__ void softmax_scale_backward_kernel_large(float *dGradAttnScore, float *dAttnScore, float *dGradAttnScale,
                                            float *dGradAttn, int seq_len, int num_batches, float scale, int tmp)
{
    //int i = blockIdx.x * seq_len * seq_len + (threadIdx.x * tmp + idx);
    for(int t = 0; t<tmp; t++){
        int i = blockIdx.x * seq_len * seq_len + (t * blockDim.x) + threadIdx.x;
        if (i >= seq_len*seq_len*num_batches)
            return;
        //printf("%d\n",i);
        for (int k = 0; k < seq_len; k++){
            dGradAttnScore[k * seq_len + i] = dGradAttnScore[k * seq_len + i] * dAttnScore[k * seq_len + i];
            //printf("%f * %f = %f\n",dGradAttnScore[k * seq_len + i],dAttnScore[k * seq_len + i],dGradAttnScoreScale[k * seq_len + i]);
        }

        float sum = 0.0f;
        for (int k = 0; k < seq_len; k++){
            sum += dGradAttnScore[k * seq_len + i];
        }
        //printf("%f\n",sum);
        for (int k = 0; k < seq_len; k++){
            dGradAttn[k * seq_len + i] = (dGradAttnScore[k * seq_len + i] - (dAttnScore[k * seq_len + i] * sum)) * scale;
        }
    }
}

void gemm_strided_batchedEx(hipblasHandle_t handle, hipblasOperation_t opA, const float *d_A, const float *d_B, float *d_C, const int M, const int N, const int K, const int batch_size) {

    int lda = M;
    int ldb = K;
    int ldc = M;

    int strideA = M*K;
    int strideB = K*N;
    int strideC = M*N;

    // Set the alpha and beta parameters for the gemm operation
    float alpha = 1.0f;
    float beta = 0.0f;

    // Perform the matrix multiplication using cublasGemmStridedBatchedEx
    hipblasGemmStridedBatchedEx(handle,
                               opA, HIPBLAS_OP_N,
                               M, N, K,
                               &alpha,
                               d_A, HIP_R_32F, lda, strideA,
                               d_B, HIP_R_32F, ldb, strideB,
                               &beta,
                               d_C, HIP_R_32F, ldc, strideC,
                               batch_size,
                               HIPBLAS_COMPUTE_32F_FAST_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
 
}
//CUBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT
//CUBLAS_COMPUTE_32F_FAST_16F, CUBLAS_GEMM_DEFAULT_TENSOR_OP
struct Attention{
    hipblasHandle_t handle;
    hipEvent_t start, stop;
    float milliseconds = 0;
    float scale = 0;
    int emb_dim, seq_len, num_batches, tmp_seq_len, tmp_num_batches, tmp;

    Attention(int hemb_dim, int hseq_len, int hnum_batches){
        hipblasCreate(&handle);
        hipEventCreate(&start);
        hipEventCreate(&stop);
        
        emb_dim = hemb_dim;
        seq_len = hseq_len;
        num_batches = hnum_batches;
        tmp_seq_len = 1024;

        if(seq_len>tmp_seq_len){
            tmp = (int)(seq_len/tmp_seq_len);
            tmp_num_batches = num_batches * tmp;
            
        }
        else{
            tmp_seq_len = seq_len;
            tmp_num_batches = num_batches;
            tmp = 0;
        }

        scale = 1/sqrtf(float(emb_dim));
    }

    ~Attention(){
        hipblasDestroy(handle);

    }

    void attn_forward(float *dQuery, float *dKey, float *dValue, float *dAttn, float *dOut)
    {
        // Host problem definition
        //cudaEventRecord(start);
        gemm_strided_batchedEx(handle, HIPBLAS_OP_N, dQuery, dKey, dAttn, seq_len, seq_len, emb_dim, num_batches);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("Forward gemm_strided_batchedEx 1 : %f\n",milliseconds);

        // cudaEventRecord(start);
        //scale_kernel<<<num_batches, seq_len*seq_len>>>(dAttn, seq_len, num_batches, scale);
        //cudaDeviceSynchronize();
        
        if(tmp == 0){
            max_sum_exp_kernel<<<num_batches, seq_len>>>(dAttn, seq_len, scale, num_batches);
        }
        else{
            max_sum_exp_kernel_large<<<num_batches, tmp_seq_len>>>(dAttn, seq_len, scale, num_batches, tmp);
        }
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("Forward scale_softmax_function : %f\n",milliseconds);

        // cudaEventRecord(start);
        gemm_strided_batchedEx(handle, HIPBLAS_OP_N, dAttn, dValue, dOut, seq_len, emb_dim, seq_len, num_batches);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("Forward gemm_strided_batchedEx 2 : %f\n",milliseconds);
    }

    void attn_backward(float *dQuery, float *dKey, float *dValue, float *dAttnScore, float *dGradOutput, float *dGradAttnScore, 
                    float *dGradAttnScale, float *dGradAttn, float *dGradQuery, float *dGradKey, float *dGradValue)
    {
        //cudaEventRecord(start);
        gemm_strided_batchedEx(handle, HIPBLAS_OP_N, dGradOutput, dValue, dGradAttnScore, seq_len, seq_len, emb_dim, num_batches);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("Backward gemm_strided_batchedEx 1: %f\n",milliseconds);

        // cudaEventRecord(start);
        if(tmp == 0){
            softmax_scale_backward_kernel<<<num_batches, seq_len>>>(dGradAttnScore, dAttnScore, dGradAttnScale, dGradAttn, seq_len, emb_dim, num_batches, scale);
        }
        else{
            softmax_scale_backward_kernel_large<<<num_batches, tmp_seq_len>>>(dGradAttnScore, dAttnScore, dGradAttnScale, dGradAttn, seq_len, num_batches, scale, tmp);
        }
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("Backward softmax_scale_backward_function 2: %f\n",milliseconds);

        // cudaEventRecord(start);
        gemm_strided_batchedEx(handle, HIPBLAS_OP_N, dGradAttn, dKey, dGradQuery, seq_len, emb_dim, seq_len, num_batches);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("Backward gemm_strided_batchedEx 2: %f\n",milliseconds);

        // cudaEventRecord(start);
        gemm_strided_batchedEx(handle, HIPBLAS_OP_T, dGradAttn, dQuery, dGradKey, seq_len, emb_dim, seq_len, num_batches);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("Backward gemm_strided_batchedEx 3: %f\n",milliseconds);

        // cudaEventRecord(start);
        gemm_strided_batchedEx(handle, HIPBLAS_OP_T, dAttnScore, dGradOutput, dGradValue, seq_len, emb_dim, seq_len, num_batches);
        // cudaEventRecord(stop);
        // cudaEventSynchronize(stop);
        // milliseconds = 0;
        // cudaEventElapsedTime(&milliseconds, start, stop);
        // printf("Backward gemm_strided_batchedEx 4: %f\n",milliseconds);
    }
};

extern "C" Attention* init(int emb_dim, int seq_len, int num_batches) {
    return new Attention(emb_dim, seq_len, num_batches);
}

extern "C" void attn_forward(Attention* attn, float *hQuery, float *hKey, float *hValue, float *hAttn, float *hOut){
    attn->attn_forward(hQuery, hKey, hValue, hAttn, hOut);
}

extern "C" void attn_backward(Attention* attn, float *hQuery, float *hKey, float *hValue, float *hAttnScore, float *hGradOutput, float *hGradAttnScore, 
                    float *hGradAttnScale, float *hGradAttn, float *hGradQuery, float *hGradKey, float *hGradValue){
    attn->attn_backward(hQuery, hKey, hValue, hAttnScore, hGradOutput, hGradAttnScore, hGradAttnScale, hGradAttn, hGradQuery, hGradKey, hGradValue);
}

extern "C" void destroy(Attention* attn) {
    delete attn;
}
